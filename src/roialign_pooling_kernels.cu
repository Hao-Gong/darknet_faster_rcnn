#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <float.h>
// implementation taken from Caffe2
extern "C" {
#include "hip/hip_runtime.h"
#include "roialign_pooling.h"
}

// TODO make it in a common file
// #define CUDA_1D_KERNEL_LOOP(i, n)                            \
//   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
//        i += blockDim.x * gridDim.x)



// __device__ float bilinear_interpolate(const float* bottom_data,
//     const int height, const int width,
//     float y, float x,
//     const int index /* index for debug only*/) {

//   // deal with cases that inverse elements are out of feature map boundary
//   if (y < -1.0 || y > height || x < -1.0 || x > width) {
//     //empty
//     return 0;
//   }

//   if (y <= 0) y = 0;
//   if (x <= 0) x = 0;

//   int y_low = (int) y;
//   int x_low = (int) x;
//   int y_high;
//   int x_high;

//   if (y_low >= height - 1) {
//     y_high = y_low = height - 1;
//     y = (float) y_low;
//   } else {
//     y_high = y_low + 1;
//   }

//   if (x_low >= width - 1) {
//     x_high = x_low = width - 1;
//     x = (float) x_low;
//   } else {
//     x_high = x_low + 1;
//   }

//   float ly = y - y_low;
//   float lx = x - x_low;
//   float hy = 1. - ly, hx = 1. - lx;
//   // do bilinear interpolation
//   float v1 = bottom_data[y_low * width + x_low];
//   float v2 = bottom_data[y_low * width + x_high];
//   float v3 = bottom_data[y_high * width + x_low];
//   float v4 = bottom_data[y_high * width + x_high];
//   float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

//   float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

//   return val;
// }

// __global__ void RoIAlignForward(const int nthreads, const float* bottom_data,
//     const float spatial_scale, const int channels,
//     const int height, const int width,
//     const int pooled_height, const int pooled_width,
//     const int sampling_ratio,
//     const float* bottom_rois, float* top_data) {
//   CUDA_1D_KERNEL_LOOP(index, nthreads) {
//     // (n, c, ph, pw) is an element in the pooled output
//     int pw = index % pooled_width;
//     int ph = (index / pooled_width) % pooled_height;
//     int c = (index / pooled_width / pooled_height) % channels;
//     int n = index / pooled_width / pooled_height / channels;

//     const float* offset_bottom_rois = bottom_rois + n * 5;
//     int roi_batch_ind = offset_bottom_rois[0];

//     // Do not using rounding; this implementation detail is critical
//     float roi_start_w = offset_bottom_rois[1] * spatial_scale;
//     float roi_start_h = offset_bottom_rois[2] * spatial_scale;
//     float roi_end_w = offset_bottom_rois[3] * spatial_scale;
//     float roi_end_h = offset_bottom_rois[4] * spatial_scale;
//     // float roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
//     // float roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
//     // float roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
//     // float roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

//     // Force malformed ROIs to be 1x1
//     float roi_width = max(roi_end_w - roi_start_w, (float)1.);
//     float roi_height = max(roi_end_h - roi_start_h, (float)1.);
//     float bin_size_h = (float)(roi_height) / (float)(pooled_height);
//     float bin_size_w = (float)(roi_width) / (float)(pooled_width);

//     const float* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

//     // We use roi_bin_grid to sample the grid and mimic integral
//     int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
//     int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

//     // We do average (integral) pooling inside a bin
//     const float count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

//     float output_val = 0.;
//     for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
//     {
//       const float y = roi_start_h + ph * bin_size_h + (float)(iy + .5f) * bin_size_h / (float)(roi_bin_grid_h); // e.g., 0.5, 1.5
//       for (int ix = 0; ix < roi_bin_grid_w; ix ++)
//       {
//         const float x = roi_start_w + pw * bin_size_w + (float)(ix + .5f) * bin_size_w / (float)(roi_bin_grid_w);

//         float val = bilinear_interpolate(offset_bottom_data, height, width, y, x, index);
//         output_val += val;
//       }
//     }
//     output_val /= count;

//     top_data[index] = output_val;
//   }
// }

// __device__ void bilinear_interpolate_gradient(
//     const int height, const int width,
//     float y, float x,
//     float & w1, float & w2, float & w3, float & w4,
//     int & x_low, int & x_high, int & y_low, int & y_high,
//     const int index /* index for debug only*/) {

//     // deal with cases that inverse elements are out of feature map boundary
//     if (y < -1.0 || y > height || x < -1.0 || x > width) {
//       //empty
//       w1 = w2 = w3 = w4 = 0.;
//       x_low = x_high = y_low = y_high = -1;
//       return;
//     }

//     if (y <= 0) y = 0;
//     if (x <= 0) x = 0;

//     y_low = (int) y;
//     x_low = (int) x;

//     if (y_low >= height - 1) {
//       y_high = y_low = height - 1;
//       y = (float) y_low;
//     } else {
//       y_high = y_low + 1;
//     }

//     if (x_low >= width - 1) {
//       x_high = x_low = width - 1;
//       x = (float) x_low;
//     } else {
//       x_high = x_low + 1;
//     }

//     float ly = y - y_low;
//     float lx = x - x_low;
//     float hy = 1. - ly, hx = 1. - lx;

//     // reference in forward
//     // float v1 = bottom_data[y_low * width + x_low];
//     // float v2 = bottom_data[y_low * width + x_high];
//     // float v3 = bottom_data[y_high * width + x_low];
//     // float v4 = bottom_data[y_high * width + x_high];
//     // float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

//     w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

//     return;
// }

// __global__ void RoIAlignBackwardFeature(const int nthreads, const float* top_diff,
//     const int num_rois, const float spatial_scale,
//     const int channels, const int height, const int width,
//     const int pooled_height, const int pooled_width,
//     const int sampling_ratio,
//     float* bottom_diff,
//     const float* bottom_rois) {
//   CUDA_1D_KERNEL_LOOP(index, nthreads) {
//     // (n, c, ph, pw) is an element in the pooled output
//     int pw = index % pooled_width;
//     int ph = (index / pooled_width) % pooled_height;
//     int c = (index / pooled_width / pooled_height) % channels;
//     int n = index / pooled_width / pooled_height / channels;

//     const float* offset_bottom_rois = bottom_rois + n * 5;
//     int roi_batch_ind = offset_bottom_rois[0];

//     // Do not using rounding; this implementation detail is critical
//     float roi_start_w = offset_bottom_rois[1] * spatial_scale;
//     float roi_start_h = offset_bottom_rois[2] * spatial_scale;
//     float roi_end_w = offset_bottom_rois[3] * spatial_scale;
//     float roi_end_h = offset_bottom_rois[4] * spatial_scale;
//     // float roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
//     // float roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
//     // float roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
//     // float roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

//     // Force malformed ROIs to be 1x1
//     float roi_width = max(roi_end_w - roi_start_w, (float)1.);
//     float roi_height = max(roi_end_h - roi_start_h, (float)1.);
//     float bin_size_h = (float)(roi_height) / (float)(pooled_height);
//     float bin_size_w = (float)(roi_width) / (float)(pooled_width);

//     float* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;

//     int top_offset    = (n * channels + c) * pooled_height * pooled_width;
//     const float* offset_top_diff = top_diff + top_offset;
//     const float top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

//     // We use roi_bin_grid to sample the grid and mimic integral
//     int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
//     int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

//     // We do average (integral) pooling inside a bin
//     const float count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

//     for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
//     {
//       const float y = roi_start_h + ph * bin_size_h + (float)(iy + .5f) * bin_size_h / (float)(roi_bin_grid_h); // e.g., 0.5, 1.5
//       for (int ix = 0; ix < roi_bin_grid_w; ix ++)
//       {
//         const float x = roi_start_w + pw * bin_size_w + (float)(ix + .5f) * bin_size_w / (float)(roi_bin_grid_w);

//         float w1, w2, w3, w4;
//         int x_low, x_high, y_low, y_high;

//         bilinear_interpolate_gradient(height, width, y, x,
//             w1, w2, w3, w4,
//             x_low, x_high, y_low, y_high,
//             index);

//         float g1 = top_diff_this_bin * w1 / count;
//         float g2 = top_diff_this_bin * w2 / count;
//         float g3 = top_diff_this_bin * w3 / count;
//         float g4 = top_diff_this_bin * w4 / count;

//         if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0)
//         {
//           atomicAdd(offset_bottom_diff + y_low * width + x_low, (float)(g1));
//           atomicAdd(offset_bottom_diff + y_low * width + x_high, (float)(g2));
//           atomicAdd(offset_bottom_diff + y_high * width + x_low, (float)(g3));
//           atomicAdd(offset_bottom_diff + y_high * width + x_high, (float)(g4));
//         } // if
//       } // ix
//     } // iy
//   } // CUDA_1D_KERNEL_LOOP
// } // RoIAlignBackward


// 	int ROIAlignForwardLaucher(const float* bottom_data, const float spatial_scale, const int num_rois, const int height, const int width,
// 		const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* top_data,const int sampling_ratio) {
// 		const int kThreadsPerBlock = 1024;   //每个线程块（block）设置1024个线程
// 		const int output_size = num_rois * aligned_height * aligned_width * channels;   //要处理的总任务数量，即pooling完之后featuremap的大小
// 		hipError_t err;
//     hipStream_t stream = 0;
//     // dim3 grid(min(THCCeilDiv(output_size, 512L), 4096L));
//     // dim3 block(512);
// 		// 设置(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock 线程块（block）
// 		RoIAlignForward << <(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream >> >(   
// 			output_size, bottom_data, spatial_scale, height, width, channels,
// 			aligned_height, aligned_width, sampling_ratio,bottom_rois, top_data); //开始cuda

// 		err = hipGetLastError();
// 		if (hipSuccess != err) {
// 			fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
// 			exit(-1);
// 		}
// 		return 1;
// 	}

// 	int ROIAlignBackwardLaucher(const float* top_diff, const float spatial_scale, const int num_rois, const int height, const int width,
// 		const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* bottom_diff,const int sampling_ratio) {
// 		const int kThreadsPerBlock = 1024;
//     hipStream_t stream = 0;
// 		const int output_size = num_rois * aligned_height * aligned_width * channels;
// 		hipError_t err;

//     RoIAlignBackwardFeature<< <(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream >> >(
//         output_size, 
//         top_diff, 
//         num_rois,
//         spatial_scale,
//         channels,
//         height,
//         width,
//         aligned_height, 
//         aligned_width, 
//         sampling_ratio,
//         bottom_diff, bottom_rois);

// 		err = hipGetLastError();
// 		if (hipSuccess != err) {
// 			fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
// 			exit(-1);
// 		}

// 		return 1;
// 	}

// TODO remove the dependency on input and use instead its sizes -> save memory

// 当前开辟的所有线程数是blockDim.x * gridDim.x，当需要并行的任务总数超过了当前开辟的所有线程数时，可以让线程循环的完成任务。一种常见的用法。
// 比如，一共开辟了5*2共十个线程，一共有30个任务，0号线程在干完任务0后，可以继续干任务0+10，之后可以继续干任务0+10+10。同理1号线程可以按顺序去做任务1,11,21。
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
            i += blockDim.x * gridDim.x)             


	__global__ void ROIAlignForward(const int nthreads, const float* bottom_data, const float spatial_scale, const int height, const int width,
		const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* top_data) {
		/*
		nthreads：pooling后的featuremap像素点总数量，即num_rois * aligned_height * aligned_width * channels，num_rois表示当前batch里所有的roi数量，
		aligned_height，aligned_width分别表示pooling后的h和pooling后的w，channels表示通道数（pooling前后不变）。每个线程负责一个pooling结果，所以这个数值也是线程总数量
		bottom_data：需要进行roialign的featuremap的首地址，注意这个特征图由python里的(bs, c, h, w)4维矩阵变成了c语言里的(bs*c*h*w)一维数组。
		spatial_scale：原图和特诊图之间的比例。原图的height/特征图的height
		height：特征图的height
		width：特征图的width
		channels：特征图的channels
		aligned_height：pooling后的h,一般为7
		aligned_width：pooling后的w，一般为7
		bottom_rois：存储rois的首地址，在python里是2维的(num_rois, 5)，[[batch_index,x1,y1,x2,y2],...]，这里变成了c语言里的(num_rois * 5)一维数组。
		top_data：pooling结果的首地址，最后的结果存储在这里。它的形状是(num_rois * aligned_height * aligned_width * channels)一维数组，每一个都和index对应
		*/
		CUDA_1D_KERNEL_LOOP(index, nthreads) {  // 用函数宏定义中的内容代替，即index代替for循环中的i,nthreads代替for循环中的n

			// (n, c, ph, pw) is an element in the aligned output
			/*
			根据index（线程号）判断，当前线程应该计算top_data的哪个位置，
			当前计算的就是第n个roi中的第c个通道上的ph（取值范围:[0, aligned_height)）,pw（取值范围:[0, aligned_width)）块
			*/

			int pw = index % aligned_width;
			int ph = (index / aligned_width) % aligned_height;
			int c = (index / aligned_width / aligned_height) % channels;
			int n = index / aligned_width / aligned_height / channels;

			// bottom_rois += n * 5;
			float roi_batch_ind = bottom_rois[n * 5 + 0];     // bottom_rois以5位单位，0位置放当前roi属于当前batch中的第几张图片(从0开始排序)，也就是batch_index
			float roi_start_w = bottom_rois[n * 5 + 1] * spatial_scale;  // 1-4位置放当前roi左上角，右下角坐标
			float roi_start_h = bottom_rois[n * 5 + 2] * spatial_scale; //这些坐标是在featuremap上的坐标，通过spatial_scale转换过来，注意是float类型，无损失!!!
			float roi_end_w = bottom_rois[n * 5 + 3] * spatial_scale;
			float roi_end_h = bottom_rois[n * 5 + 4] * spatial_scale;

			// Force malformed ROIs to be 1x1
			float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.); // roi区域宽度，注意是float类型，无损失!!!
			float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.); //roi区域高度，注意是float类型，无损失!!!
			float bin_size_h = roi_height / (aligned_height - 1.); //这个地方是这份代码的特别之处，和原版的roialign有个小不同。他把roi区域分成了(aligned_height - 1.)*(aligned_width - 1.)个块，
			float bin_size_w = roi_width / (aligned_width - 1.); //那么在height方向可以产生aligned_height个交点，在width方向可以产生aligned_width个交点，后面就是用双线性插值求交点处的值
																
			float h = (float)(ph)* bin_size_h + roi_start_h; //当前所求的块（交点）处的h坐标，注意是float类型，无损失!!!
			float w = (float)(pw)* bin_size_w + roi_start_w; //当前所求的块（交点）处的w坐标，注意是float类型，无损失!!!

			int hstart = fminf(floor(h), height - 2); // 获得双线性插值采样点（交点）周围四个坐标中的左上角坐标。注意是int类型，准备双插!!!
			int wstart = fminf(floor(w), width - 2);  //之所以和width-2比较取较小值，是因为现在求的是左上角，要给右下角留下位置，不能让右下角超出featuremap范围   

			int img_start = roi_batch_ind * channels * height * width; //当前处理featuremap在bottom_data中的起始位置。bottom_data是一维的，所以每一个featuremap占据channels * height * width位置

			// bilinear interpolation
			if (h < 0 || h >= height || w < 0 || w >= width) { //超出featuremap范围的交点直接置0
				top_data[index] = 0.;
			}
			else {
				float h_ratio = h - (float)(hstart);
				float w_ratio = w - (float)(wstart);
				int upleft = img_start + (c * height + hstart) * width + wstart;  //把左上角左边从3维度变成一维度。因为bottom_data是一维度的
				int upright = upleft + 1;
				int downleft = upleft + width;   //左下角坐标和左上角坐标在一维度上相差width
				int downright = downleft + 1;

				top_data[index] = bottom_data[upleft] * (1. - h_ratio) * (1. - w_ratio)
					+ bottom_data[upright] * (1. - h_ratio) * w_ratio
					+ bottom_data[downleft] * h_ratio * (1. - w_ratio)
					+ bottom_data[downright] * h_ratio * w_ratio;       //双线性插值公式 f(i+u,j+v) = (1-u)(1-v)f(i,j)+ u(1-v)f(i+1,j) + (1-u)vf(i,j+1) + uvf(i+1,j+1)
			}
		}
	}


	int ROIAlignForwardLaucher(const float* bottom_data, const float spatial_scale, const int num_rois, const int height, const int width,
		const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* top_data, hipStream_t stream) {
		const int kThreadsPerBlock = 1024;   //每个线程块（block）设置1024个线程
		const int output_size = num_rois * aligned_height * aligned_width * channels;   //要处理的总任务数量，即pooling完之后featuremap的大小
		hipError_t err;

		// 设置(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock 线程块（block）
		ROIAlignForward << <(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream >> >(   
			output_size, bottom_data, spatial_scale, height, width, channels,
			aligned_height, aligned_width, bottom_rois, top_data); //开始cuda

		err = hipGetLastError();
		if (hipSuccess != err) {
			fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
			exit(-1);
		}

		return 1;
	}


	__global__ void ROIAlignBackward(const int nthreads, const float* top_diff, const float spatial_scale, const int height, const int width,
		const int channels, const int aligned_height, const int aligned_width, float* bottom_diff, const float* bottom_rois) {
		/*
		roialign的反向很简单，就是正向传播时，只有参与过双插的点才会有梯度，其他的点没有梯度。所以只要把正向传播再进行一遍，找到做双插的点，每个做双插的点，在双插
		时前面的系数乘以梯度就是该点最终的梯度。
		nthreads ：任务数，和正向传播时数量一样。正向时，每个线程通过交点周围4个点的值计算交点处的值；反向时，每个线程计算一个交点周围4个点的梯度。
		top_diff ：pooling后每个点的梯度。这是存储数组的首地址。
		bottom_diff ：pooling前整个featuremap上每个点的梯度。这也是首地址，是我们想要的结果。
		*/

		CUDA_1D_KERNEL_LOOP(index, nthreads) {

			// (n, c, ph, pw) is an element in the aligned output
			int pw = index % aligned_width;
			int ph = (index / aligned_width) % aligned_height;
			int c = (index / aligned_width / aligned_height) % channels;
			int n = index / aligned_width / aligned_height / channels;

			float roi_batch_ind = bottom_rois[n * 5 + 0];
			float roi_start_w = bottom_rois[n * 5 + 1] * spatial_scale;
			float roi_start_h = bottom_rois[n * 5 + 2] * spatial_scale;
			float roi_end_w = bottom_rois[n * 5 + 3] * spatial_scale;
			float roi_end_h = bottom_rois[n * 5 + 4] * spatial_scale;
			/* int roi_start_w = round(bottom_rois[1] * spatial_scale); */
			/* int roi_start_h = round(bottom_rois[2] * spatial_scale); */
			/* int roi_end_w = round(bottom_rois[3] * spatial_scale); */
			/* int roi_end_h = round(bottom_rois[4] * spatial_scale); */

			// Force malformed ROIs to be 1x1
			float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.);
			float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.);
			float bin_size_h = roi_height / (aligned_height - 1.);
			float bin_size_w = roi_width / (aligned_width - 1.);
    // We use roi_bin_grid to sample the grid and mimic integral
    // int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    // int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);
			float h = (float)(ph)* bin_size_h + roi_start_h;
			float w = (float)(pw)* bin_size_w + roi_start_w;

			int hstart = fminf(floor(h), height - 2);
			int wstart = fminf(floor(w), width - 2);

			int img_start = roi_batch_ind * channels * height * width;

			// bilinear interpolation
			if (!(h < 0 || h >= height || w < 0 || w >= width)) {
				float h_ratio = h - (float)(hstart);
				float w_ratio = w - (float)(wstart);
				int upleft = img_start + (c * height + hstart) * width + wstart;
				int upright = upleft + 1;
				int downleft = upleft + width;
				int downright = downleft + 1;

				//以上都和前向传播一样，下面是计算4个点的梯度，双插系数*梯度
				atomicAdd(bottom_diff + upleft, top_diff[index] * (1. - h_ratio) * (1 - w_ratio));
				atomicAdd(bottom_diff + upright, top_diff[index] * (1. - h_ratio) * w_ratio);
				atomicAdd(bottom_diff + downleft, top_diff[index] * h_ratio * (1 - w_ratio));
				atomicAdd(bottom_diff + downright, top_diff[index] * h_ratio * w_ratio);
			}
		}
	}

	int ROIAlignBackwardLaucher(const float* top_diff, const float spatial_scale, const int num_rois, const int height, const int width,
		const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* bottom_diff, hipStream_t stream) {
		const int kThreadsPerBlock = 1024;
		const int output_size = num_rois * aligned_height * aligned_width * channels;
		hipError_t err;

		ROIAlignBackward << <(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream >> >(
			output_size, top_diff, spatial_scale, height, width, channels,
			aligned_height, aligned_width, bottom_diff, bottom_rois);

		err = hipGetLastError();
		if (hipSuccess != err) {
			fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
			exit(-1);
		}

		return 1;
	}
